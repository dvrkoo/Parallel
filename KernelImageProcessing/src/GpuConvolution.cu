#include "hip/hip_runtime.h"
#include "GpuConvolution.h"
#include <iostream>

#define checkCuda(val) check((val), #val, __FILE__, __LINE__)
static void check(hipError_t result, const char* func, const char* file, int line) {
    if (result) {
        std::cerr << "CUDA error=" << static_cast<int>(result) << " (" << hipGetErrorString(result) << ") at " << file << ":" << line << " '" << func << "'\n";
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}

__global__ void conv2dKernel(const unsigned char* in, float* out,
                             int width, int height, int channels,
                             const float* kernel, int kW, int kH) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int cx = kW / 2;
    int cy = kH / 2;
    if (x < width && y < height) {
        for (int c = 0; c < channels; ++c) {
            float sum = 0;
            for (int m = 0; m < kH; ++m) {
                int yy = y + m - cy;
                if (yy < 0 || yy >= height) continue;
                for (int n = 0; n < kW; ++n) {
                    int xx = x + n - cx;
                    if (xx < 0 || xx >= width) continue;
                    int neighbor_idx = (yy * width + xx) * channels + c;
                    sum += in[neighbor_idx] * kernel[m * kW + n];
                }
            }
            int output_idx = (y * width + x) * channels + c;
            out[output_idx] = sum;
        }
    }
}


template <typename InType, typename OutType>
__global__ void conv2dSharedMemKernel_Final(const InType* in, OutType* out,
                                            int width, int height,
                                            const float* kernel, int k_side) {
    
    extern __shared__ unsigned char tile_raw[];
    InType* const tile = (InType*)tile_raw;

    const int HALO = k_side / 2;
    // The tile width needs to accommodate the block width plus the halo on both sides.
    const int TILE_WIDTH = blockDim.x + 2 * HALO;
    // The tile height is simply the block height. We load 'k_side' rows for each block row.
    const int TILE_HEIGHT = blockDim.y;


    // --- High-Performance Loading Pattern (Corrected) ---
    // Each thread loads a vertical stripe of 'k_side' pixels.
    int base_load_x = blockIdx.x * blockDim.x + threadIdx.x - HALO;
    int base_load_y = blockIdx.y * blockDim.y - HALO;

    for (int m = 0; m < k_side; ++m) {
        int load_y = base_load_y + threadIdx.y + m;
        int tile_y = threadIdx.y + m;

        if (base_load_x >= 0 && base_load_x < width && load_y >= 0 && load_y < height) {
            // This load is coalesced because adjacent threads access adjacent memory.
            tile[tile_y * TILE_WIDTH + threadIdx.x] = in[load_y * width + base_load_x];
        } else {
            memset(&tile[tile_y * TILE_WIDTH + threadIdx.x], 0, sizeof(InType));
        }
    }
    
    // To complete the halo on the right side, the first few threads in each row
    // need to load extra pixels.
    for (int m = 0; m < k_side; ++m) {
        if (threadIdx.x < 2 * HALO) {
            int load_y = base_load_y + threadIdx.y + m;
            int tile_y = threadIdx.y + m;
            int load_x = base_load_x + blockDim.x + threadIdx.x; // Load from the right
            int tile_x = blockDim.x + threadIdx.x;

            if (load_x >= 0 && load_x < width && load_y >= 0 && load_y < height) {
                tile[tile_y * TILE_WIDTH + tile_x] = in[load_y * width + load_x];
            } else {
                memset(&tile[tile_y * TILE_WIDTH + tile_x], 0, sizeof(InType));
            }
        }
    }

    __syncthreads();

    // --- Computation (uses the loaded tile) ---
    int out_x = blockIdx.x * blockDim.x + threadIdx.x;
    int out_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (out_x < width && out_y < height) {
        OutType sum;
        memset(&sum, 0, sizeof(OutType));

        // The thread's local (x,y) position within the shared memory tile's top-left corner
        const int local_x_in_tile = threadIdx.x;
        const int local_y_in_tile = threadIdx.y;

        for (int m = 0; m < k_side; m++) {
            for (int n = 0; n < k_side; n++) {
                // Read from the sub-region relative to the thread's position
                InType pixel = tile[(local_y_in_tile + m) * TILE_WIDTH + (local_x_in_tile + n)];
                float k_val = kernel[m * k_side + n];

                if constexpr (sizeof(InType) == 1) { // Grayscale
                    sum += pixel * k_val;
                } else { // Color
                    sum.x += pixel.x * k_val;
                    sum.y += pixel.y * k_val;
                    sum.z += pixel.z * k_val;
                }
            }
        }
        out[out_y * width + out_x] = sum;
    }
}


GpuConvolution::GpuConvolution(const cv::Mat& kernel, bool use_shared_memory)
    : kernel_(kernel), kRows_(kernel.rows), kCols_(kernel.cols), use_shared_mem_(use_shared_memory) 
{
    // The rest of the constructor body is the same.
    CV_Assert(kernel.type() == CV_32F);
    
    size_t kernBytes = kRows_ * kCols_ * sizeof(float);
    checkCuda(hipMalloc(&d_k_, kernBytes));
    checkCuda(hipMemcpy(d_k_, kernel_.ptr<float>(), kernBytes, hipMemcpyHostToDevice));
}

GpuConvolution::~GpuConvolution() {
    if (d_k_) {
        hipFree(d_k_);
    }
}


// In GpuConvolution.cu

cv::Mat GpuConvolution::apply(const cv::Mat& input, const dim3& blockDim, int maxGridDimX) {
    int w = input.cols;
    int h = input.rows;
    int channels = input.channels();

    // (Memory allocation and setup logic is the same...)
    size_t inputBytes = w * h * channels * sizeof(unsigned char);
    size_t outputBytes = w * h * channels * sizeof(float);
    unsigned char* d_in;
    float* d_out;
    checkCuda(hipMalloc(&d_in, inputBytes));
    checkCuda(hipMalloc(&d_out, outputBytes));
    checkCuda(hipMemcpy(d_in, input.data, inputBytes, hipMemcpyHostToDevice));

    dim3 gridDim((w + blockDim.x - 1) / blockDim.x, (h + blockDim.y - 1) / blockDim.y);
    if (maxGridDimX > 0 && maxGridDimX < gridDim.x) {
        gridDim.x = maxGridDimX;
    }

    if (use_shared_mem_) {
        CV_Assert(blockDim.x == 16 && blockDim.y == 16 && "Shared memory kernel is optimized for 16x16 blocks.");
        
        int halo = kCols_ / 2;
        int tile_width = blockDim.x + 2 * halo;
        int tile_height = blockDim.y + 2 * halo; // <-- This was the subtle bug here
        if (channels == 3) {
            size_t shared_mem_bytes = tile_height * tile_width * sizeof(uchar3);
            conv2dSharedMemKernel_Final<uchar3, float3><<<gridDim, blockDim, shared_mem_bytes>>>(
                (const uchar3*)d_in, (float3*)d_out, w, h, d_k_, kCols_);

        } else if (channels == 1) {
            size_t shared_mem_bytes = tile_width * tile_height * sizeof(unsigned char);
            conv2dSharedMemKernel_Final<unsigned char, float><<<gridDim, blockDim, shared_mem_bytes>>>(
                (const unsigned char*)d_in, (float*)d_out, w, h, d_k_, kCols_);

        } else {
             CV_Assert(false && "Shared memory kernel only supports 1 or 3 channel images.");
        }

    } else { // Use the original naive kernel
        conv2dKernel<<<gridDim, blockDim>>>(d_in, (float*)d_out, w, h, channels, d_k_, kCols_, kRows_);
    }

    checkCuda(hipGetLastError());
    
    cv::Mat out(h, w, CV_32FC(channels));
    checkCuda(hipMemcpy(out.data, d_out, outputBytes, hipMemcpyDeviceToHost));
    
    hipFree(d_in);
    hipFree(d_out);
    
    return out;
}
